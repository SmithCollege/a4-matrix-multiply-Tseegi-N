#include <iostream>
#include "hipblas.h"
#include <hip/hip_runtime.h>

// define constants
#define M 900  // Rows of x and z
#define N 900  // Columns of y and z
#define K 900  // Columns of x and rows of y

void matrixMul(const float* x, const float* y, float* z, int m, int n, int k) {
  hipblasHandle_t handle;
  hipblasCreate(&handle);

  const float alpha = 1.0f;
  const float beta = 0.0f;
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  // Perform the matrix multiplication: z = alpha * x * y + beta * z
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, &alpha,
    x, m,
    y, k,
    &beta,
    z, m);

  hipblasDestroy(handle);
}

int main() {
  // Allocate CPU matrices
  float x[M * K], y[K * N], z[M * N] = {0};
  for (int i = 0; i < M * K; i++) x[i] = 1.0f;
  for (int i = 0; i < K * N; i++) y[i] = 1.0f;

  size_t freeMem, totalMem;
  hipMemGetInfo(&freeMem, &totalMem);
  std::cout << "Before allocation: Free memory: " << freeMem << " bytes, Total memory: " << totalMem << " bytes\n";

  // Allocate GPU matrices
  float *d_x, *d_y, *d_z;
  hipMalloc((void**)&d_x, M * K * sizeof(float));
  hipMalloc((void**)&d_y, K * N * sizeof(float));
  hipMalloc((void**)&d_z, M * N * sizeof(float));

  hipMemGetInfo(&freeMem, &totalMem);
  std::cout << "After allocation: Free memory: " << freeMem << " bytes, Total memory: " << totalMem << " bytes\n";

  // Copy matrices to GPU
  hipMemcpy(d_x, x, M * K * sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(d_y, y, K * N * sizeof(float), hipMemcpyHostToDevice);
  hipMemset(d_z, 0, M * N * sizeof(float));
  printf("%s\n", hipGetErrorString(hipGetLastError()));
  matrixMul(d_x, d_y, d_z, M, N, K);
  printf("%s\n", hipGetErrorString(hipGetLastError()));

  // Copy result back to host
  hipMemcpy(z, d_z, M * N * sizeof(float), hipMemcpyDeviceToHost);

  std::cout << "z[0]: " << z[0] << " z[N-1]: " << z[N -1] << std::endl;

  // Free memory
  hipFree(d_x);
  hipFree(d_y);
  hipFree(d_z);
  return 0;
}
